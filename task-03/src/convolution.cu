#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <algorithm>
#include <hipDNN.h>

#include "convolution.cuh"
#include "launcher.cuh"
#include "kernels.cuh"

Convolution::Convolution(hipdnnDataType_t type) : type(type) {
    CUDNN_CHECK_ERROR(hipdnnCreate(&cudnn));

    // Создание дескрипторов памяти
    CUDNN_CHECK_ERROR(hipdnnCreateTensorDescriptor(&data_descriptor));

    // Создание дескриптора под свёртку
    CUDNN_CHECK_ERROR(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CHECK_ERROR(hipdnnCreateFilterDescriptor(&filter_descriptor));
}

Convolution::~Convolution() {
    // Очистка ресурсов
    CUDNN_CHECK_ERROR(hipdnnDestroy(cudnn));

    CUDNN_CHECK_ERROR(hipdnnDestroyTensorDescriptor(data_descriptor));

    CUDNN_CHECK_ERROR(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CHECK_ERROR(hipdnnDestroyFilterDescriptor(filter_descriptor));

    if (f_workspace != nullptr) CUDA_CHECK_ERROR(hipFree(f_workspace));
    if (b_workspace != nullptr) CUDA_CHECK_ERROR(hipFree(b_workspace));

    if (input != nullptr) CUDA_CHECK_ERROR(hipFree(input));
    if (output != nullptr) CUDA_CHECK_ERROR(hipFree(output));
    if (expected != nullptr) CUDA_CHECK_ERROR(hipFree(expected));
    if (output_grad != nullptr) CUDA_CHECK_ERROR(hipFree(output_grad));
    if (d_tmp != nullptr) CUDA_CHECK_ERROR(hipFree(d_tmp));
}

bool Convolution::set_data_desc(int input_height, int input_width, int input_channels) {
    if (channels != 0 && channels != input_channels) {
        std::cout << "Data channels not equal filter channels" << std::endl;
        return false;
    }

    if (channels == 0) channels = input_channels;
    height = input_height;
    width = input_width;

    size = width * height * channels;
    bytes = size * sizeof(TData);

    CUDNN_CHECK_ERROR(hipdnnSetTensor4dDescriptor(data_descriptor,
        HIPDNN_TENSOR_NCHW, type,
        1, channels, height, width));

    return true;
}

bool Convolution::set_filter_desc(int filter_dim, int filter_channels) {
    if (channels != 0 && channels != filter_channels) {
        std::cout << "Filter channels not equal data channels" << std::endl;
        return false;
    }

    if (channels == 0) channels = filter_channels;
    dim = filter_dim;

    CUDNN_CHECK_ERROR(hipdnnSetFilter4dDescriptor(filter_descriptor,
        type, HIPDNN_TENSOR_NCHW,
        channels, channels, dim, dim));

    return true;
}

bool Convolution::set_convolution() {
    int padding = (dim - 1) / 2;
    CUDNN_CHECK_ERROR(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        padding, padding, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, type));

    int output_number, output_channels, output_height, output_width;

    CUDNN_CHECK_ERROR(hipdnnGetConvolution2dForwardOutputDim(conv_descriptor,
        data_descriptor, filter_descriptor,
        &output_number, &output_channels, &output_height, &output_width));

    if (output_channels != channels || output_height != height || output_width != width) {
        std::cout << "Output parameters not equal data parameters" << std::endl;
        return false;
    }

    // Creation best algoritm
    int alfoSize = 1;
    CUDNN_CHECK_ERROR(hipdnnFindConvolutionForwardAlgorithm(cudnn,
        data_descriptor, filter_descriptor,
        conv_descriptor, data_descriptor,
        alfoSize, &alfoSize, &algo_pref));

    // Allocation Workspace
    CUDNN_CHECK_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        data_descriptor, filter_descriptor, conv_descriptor, data_descriptor, 
        algo_pref.algo, &f_workspace_bytes));

    CUDA_CHECK_ERROR(hipMalloc(&f_workspace, f_workspace_bytes));

    set_output();

    return true;
}

bool Convolution::set_convolution_bwd() {
    // Creation best algoritm
    int alfoSize = 1;
    CUDNN_CHECK_ERROR(hipdnnFindConvolutionBackwardFilterAlgorithm(cudnn,
        data_descriptor, data_descriptor, conv_descriptor, filter_descriptor,
        alfoSize, &alfoSize, &algo_pref_bwd));

    // Allocation Workspace
    CUDNN_CHECK_ERROR(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn,
        data_descriptor, data_descriptor, conv_descriptor, filter_descriptor,
        algo_pref_bwd.algo, &b_workspace_bytes));

    CUDA_CHECK_ERROR(hipMalloc(&b_workspace, b_workspace_bytes));

    set_output_grad();
    filter.set_grad();
    CUDA_CHECK_ERROR(hipMalloc(&d_tmp, bytes));

    return true;
}

bool Convolution::set_input(const Image& h_input) {
    if (!h_input.check(height, width, channels)) {
        std::cout << "Input parameters not equal data parameters" << std::endl;
        return false;
    }

    tmp.resize(size);

    for (int ch = 0; ch < channels; ++ch) {
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                uint8_t value = h_input[(y * width + x) * channels + ch];
                tmp[ch * width * height + y * width + x] = 
                    static_cast<TData>(value) / 255.f;
            }
        }
    }

    CUDA_CHECK_ERROR(hipMalloc(&input, bytes));
    CUDA_CHECK_ERROR(hipMemcpy(input, tmp.data(), bytes, hipMemcpyHostToDevice));

    return true;
}

bool Convolution::set_expected(const Image& h_expected) {
    if (!h_expected.check(height, width, channels)) {
        std::cout << "Expected parameters not equal data parameters" << std::endl;
        return false;
    }

    tmp.resize(size);

    for (int ch = 0; ch < channels; ++ch) {
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                uint8_t value = h_expected[(y * width + x) * channels + ch];
                tmp[ch * width * height + y * width + x] =
                    static_cast<TData>(value) / 255.f;
            }
        }
    }

    CUDA_CHECK_ERROR(hipMalloc(&expected, bytes));
    CUDA_CHECK_ERROR(hipMemcpy(expected, tmp.data(), bytes, hipMemcpyHostToDevice));

    return true;
}

bool Convolution::set_output() {
    CUDA_CHECK_ERROR(hipMalloc(&output, bytes));

    return true;
}

bool Convolution::set_output_grad() {
    CUDA_CHECK_ERROR(hipMalloc(&output_grad, bytes));

    return true;
}

bool Convolution::set_filter(FilterName name) {
    return filter.generate_filter(name, dim, channels);
}

bool Convolution::apply_forward() {
    CUDNN_CHECK_ERROR(hipdnnConvolutionForward(cudnn,
        &alpha, data_descriptor, input,
        filter_descriptor, filter.get_filter(),
        conv_descriptor, algo_pref.algo,
        f_workspace, f_workspace_bytes, &beta,
        data_descriptor, output));

    return true;
}

bool Convolution::apply_backward() {
    CUDNN_CHECK_ERROR(hipdnnConvolutionBackwardFilter(cudnn,
        &alpha, data_descriptor, input,
        data_descriptor, output_grad,
        conv_descriptor, algo_pref_bwd.algo,
        b_workspace, b_workspace_bytes, &beta,
        filter_descriptor, filter.get_gradient()));

     return true;
}

bool Convolution::gradient_descent(int max_cycles) {
    apply_forward();

    TData loss = find_loss_gradient();
    TData rate;

    while (loss > epsilon && count < max_cycles) {
        apply_backward();

        rate = find_learning_rate();
        apply_filter_gradient(rate);

        apply_forward();

        loss = find_loss_gradient();

        count++;

        if (count % 1000 == 0) {
            std::cout << "Curren cycle: " << count << " Current loss: " << loss;
            std::cout << " Curren rate: " << rate << std::endl;
        }
    }

    tmp.resize(filter.get_size());
    hipMemcpy(tmp.data(), filter.get_filter(), 9 * sizeof(TData), hipMemcpyDeviceToHost);
    std::cout << "Filter : " << std::endl;
    std::cout << tmp[0] << " " << tmp[1] << " " << tmp[2] << " " << std::endl;
    std::cout << tmp[3] << " " << tmp[4] << " " << tmp[5] << " " << std::endl;
    std::cout << tmp[6] << " " << tmp[7] << " " << tmp[8] << " " << std::endl;

    return true;
}

bool Convolution::get_output(Image& h_output) {
    h_output = Image(height, width, channels);

    tmp.resize(size);
    CUDA_CHECK_ERROR(hipMemcpy(tmp.data(), output, bytes, hipMemcpyDeviceToHost));

    for (int ch = 0; ch < channels; ++ch) {
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                TData value = tmp[ch * width * height + y * width + x] * 255.f;
                value = std::min(std::max(value, 0.f), 255.f);
                h_output[(y * width + x) * channels + ch] =
                    static_cast<uint8_t>(value);
            }
        }
    }

    return true;
}

TData Convolution::find_loss_gradient() {
    launch_kernel(256, size, find_gradient, size, output, expected, output_grad);

    auto tmp_sum = output_grad;
    unsigned int task_size = size;

    while (task_size > 1) {
        task_size = launch_shared_kernel(256, task_size, find_sum, tmp_sum, task_size, d_tmp);
        tmp_sum = d_tmp;
    }

    TData h_tmp_sum;
    CUDA_CHECK_ERROR(hipMemcpy(&h_tmp_sum, d_tmp, sizeof(TData), hipMemcpyDeviceToHost));

    TData ans = h_tmp_sum / size;

    return ans;
}

TData Convolution::find_learning_rate() {
    auto tmp_sum = filter.get_gradient();
    unsigned int task_size = filter.get_size();;

    while (task_size > 1) {
        task_size = launch_shared_kernel(256, task_size, find_max, tmp_sum, task_size, d_tmp);
        tmp_sum = d_tmp;
    }

    TData h_tmp_sum;
    CUDA_CHECK_ERROR(hipMemcpy(&h_tmp_sum, d_tmp, sizeof(TData), hipMemcpyDeviceToHost));

    return rate_coef / h_tmp_sum;
}

void Convolution::apply_filter_gradient(TData learning_rate) {
    launch_kernel(256, filter.get_size(), apply_gradient, 
        filter.get_size(), filter.get_filter(), filter.get_gradient(), learning_rate);
}