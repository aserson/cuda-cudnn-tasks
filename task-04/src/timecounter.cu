#include "timecounter.cuh"

#include <iostream>

TimeCounter::TimeCounter() {
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    start();
}

TimeCounter::TimeCounter(const std::string& startMessage)
    : TimeCounter::TimeCounter() {
    std::cout << startMessage;
    start();
}

TimeCounter::~TimeCounter() {
    hipEventDestroy(_stop);
    hipEventDestroy(_start);
}

void TimeCounter::restart(const std::string& startMessage) {
    std::cout << startMessage;
    start();
}

void TimeCounter::done(const std::string& stopMessage) {
    stop();
    std::cout << stopMessage << getTime() << std::endl;
}

void TimeCounter::start() {
    hipEventRecord(_start, 0);
}

void TimeCounter::stop() {
    hipEventRecord(_stop, 0);
    hipEventSynchronize(_stop);

    hipEventElapsedTime(&time, _start, _stop);
}

float TimeCounter::getTime() {
    return time / 1000;
}
